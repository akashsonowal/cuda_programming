
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int* a, int* b, int* c){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    c[i] = a[i] + b[i];
}

__managed__ int vector_a[256], vector_b[256], vector_c[256]; // memory space specifier for both CPU and GPU

int main(){ // cpu that runs cuda kernel
    for (int i = 0; i < 256; i++){
        vector_a[i] = i;
        vector_b[i] = 256 - i;
    }

    add<<<1, 256>>>(vector_a, vector_b, vector_c); // <<<Blocks, Threads per block>>> cuda kernel call
    hipDeviceSynchronize(); // waiting for all threads to complete

    int result_sum = 0;

    for (int i = 0; i < 256; i++){
        result_sum += vector_c[i];
    }

    printf("Hi Mom! I added vectors add on GPU");
    printf("Result: sum = % d", result_sum);
}
